#include "hip/hip_runtime.h"
/*
NOTE: Run in VS using x64 platform.

TODO:

SHRINKING:
- Look into edge based vs CSR based device.
- Refactor code (multiple files)
- Ignore spaces while reading file!
- Performance improvement loading a graph
- Count vertices/edges automatically (Can be done later, use globals, need to allocate memory on gpu though in runtime)
- Load graph should be a separate method

EXPANDING:
- Force undirected (edge interconnection)
- Decrease size of char in Bridge_Edge
*/

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include "kernel.h"
#include <string.h>
#include <nvgraph.h>
#include "hip/device_functions.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <random>
#include <unordered_set>
#include <unordered_map>
#include <map>

//#define SIZE_VERTICES 281903
//#define SIZE_EDGES 2312497

//#define SIZE_VERTICES 1632803
//#define SIZE_EDGES 30622564 

#define SIZE_VERTICES 6
#define SIZE_EDGES 5

//#define SIZE_VERTICES 4039
//#define SIZE_EDGES 88234
#define MAX_THREADS 1024
#define DEFAULT_EXPANDING_SAMPLE_SIZE 0.5
#define ENABLE_DEBUG_LOG false

typedef struct Sampled_Vertices sampled_vertices;
typedef struct COO_List coo_list;
typedef struct Edge edge;
typedef struct Sampled_Graph_Version;
typedef struct Bridge_Edge;
void load_graph_from_edge_list_file(int*, int*, char*);
COO_List* load_graph_from_edge_list_file_to_coo(std::vector<int>&, std::vector<int>&, char*);
int add_vertex_as_coordinate(std::vector<int>&, std::unordered_map<int, int>&, int, int);
int get_thread_size();
int calculate_node_sampled_size(float);
int get_block_size();
Sampled_Vertices* perform_edge_based_node_sampling_step(int*, int*, float);
void print_debug_log(char*);
void print_debug_log(char*, int);
void print_coo(int*, int*);
void print_csr(int*, int*);
void sample_graph(char*, char*, float);
void convert_coo_to_csr_format(int*, int*, int*, int*);
void expand_graph(char*, char*, float);
void link_using_star_topology(Sampled_Graph_Version*, int, std::vector<Bridge_Edge>&);
void add_edge_interconnection_between_graphs(int, Sampled_Graph_Version*, Sampled_Graph_Version*, std::vector<Bridge_Edge>&);
int select_random_bridge_vertex(Sampled_Graph_Version*);
void write_expanded_output_to_file(Sampled_Graph_Version*, int, std::vector<Bridge_Edge>&, char*);
void write_output_to_file(std::vector<Edge>&, char* output_path);
void check(nvgraphStatus_t);

typedef struct COO_List {
	int* source;
	int* destination;
} COO_List;

typedef struct Sampled_Vertices {
	int* vertices;
	int sampled_vertices_size;
} Sampled_Vertices;

typedef struct Edge {
	int source, destination;
} Edge;

typedef struct Sampled_Graph_Version {
	std::vector<Edge> edges;
	char label;
} Sampled_Graph_Version;

typedef struct Bridge_Edge {
	char source[20];
	char destination[20];
} Bridge_Edge;

__device__ Edge edge_data[SIZE_EDGES];
__device__ int d_edge_count = 0;

__device__ int push_edge(Edge &edge) {
	int edge_index = atomicAdd(&d_edge_count, 1);
	if (edge_index < SIZE_EDGES) {
		edge_data[edge_index] = edge;
		return edge_index;
	} else {
		printf("Maximum edge size threshold reached.");
		return -1;
	}
}

__global__
void perform_induction_step(int* sampled_vertices, int* offsets, int* indices) {
	int neighbor_index_start_offset = blockIdx.x * blockDim.x + threadIdx.x;
	int neighbor_index_end_offset = neighbor_index_start_offset + 1;

	for (int n = offsets[neighbor_index_start_offset]; n < offsets[neighbor_index_end_offset]; n++) {
		if (sampled_vertices[neighbor_index_start_offset] && sampled_vertices[indices[n]]) {
			//printf("\nAdd edge: (%d,%d).", neighbor_index_start_offset, indices[n]);
			Edge edge;
			edge.source = neighbor_index_start_offset;
			edge.destination = indices[n];
			push_edge(edge);
		}
	}
}

//__device__ Edge edge_data_expanding[SIZE_EDGES];
//__device__ int d_edge_count_expanding = 0;

__device__ int push_edge_expanding(Edge &edge, Edge* edge_data_expanding, int* d_edge_count_expanding) {
	int edge_index = atomicAdd(d_edge_count_expanding, 1);
	if (edge_index < SIZE_EDGES) {
		edge_data_expanding[edge_index] = edge;
		return edge_index;
	}
	else {
		printf("Maximum edge size threshold reached.");
		return -1;
	}
}

__global__
void perform_induction_step_expanding(int* sampled_vertices, int* offsets, int* indices, Edge* edge_data_expanding, int* d_edge_count_expanding) {
	int neighbor_index_start_offset = blockIdx.x * blockDim.x + threadIdx.x;
	int neighbor_index_end_offset = neighbor_index_start_offset + 1;

	for (int n = offsets[neighbor_index_start_offset]; n < offsets[neighbor_index_end_offset]; n++) {
		if (sampled_vertices[neighbor_index_start_offset] && sampled_vertices[indices[n]]) {
			//printf("\nAdd edge: (%d,%d).", neighbor_index_start_offset, indices[n]);
			Edge edge;
			edge.source = neighbor_index_start_offset;
			edge.destination = indices[n];
			push_edge_expanding(edge, edge_data_expanding, d_edge_count_expanding);
		}
	}
}

/*
TODO: Allocate the memory on the GPU only when you need it, after collecting the edge-based node step.
*/
int main() {
	//char* input_path = "C:\\Users\\AJ\\Documents\\example_graph.txt";
	//char* input_path = "C:\\Users\\AJ\\Desktop\\nvgraphtest\\nvGraphExample-master\\nvGraphExample\\web-Stanford.txt";
	//char* input_path = "C:\\Users\\AJ\\Desktop\\nvgraphtest\\nvGraphExample-master\\nvGraphExample\\web-Stanford_large.txt";
	char* input_path = "C:\\Users\\AJ\\Desktop\\edge_list_example.txt";
	//char* input_path = "C:\\Users\\AJ\\Desktop\\roadnet.txt";
	//char* input_path = "C:\\Users\\AJ\\Desktop\\new_datasets\\facebook_graph.txt";
	//char* input_path = "C:\\Users\\AJ\\Desktop\\output_test\\social\\soc-pokec-relationships.txt";
	//char* input_path = "C:\\Users\\AJ\\Desktop\\new_datasets\\roadNet-PA.txt";
	//char* input_path = "C:\\Users\\AJ\\Desktop\\new_datasets\\soc-pokec-relationships.txt";

	char* output_path = "C:\\Users\\AJ\\Desktop\\new_datasets\\output\\debug_small_graph.txt";

	expand_graph(input_path, output_path, 3);

	//sample_graph(input_path, output_path, 0.5);

	return 0;
}

void sample_graph(char* input_path, char* output_path, float fraction) {
	std::vector<int> source_vertices;
	std::vector<int> destination_vertices;
	COO_List* coo_list = load_graph_from_edge_list_file_to_coo(source_vertices, destination_vertices, input_path);

	// print_coo(source_vertices, target_vertices);

	// Convert the COO graph into a CSR format for the in memory GPU representation
	int* h_offsets = (int*)malloc((SIZE_VERTICES + 1) * sizeof(int));
	int* h_indices = (int*)malloc(SIZE_EDGES * sizeof(int));

	convert_coo_to_csr_format(coo_list->source, coo_list->destination, h_offsets, h_indices);

	//print_csr(h_offsets, h_indices);

	// Edge based Node Sampling Step
	Sampled_Vertices* sampled_vertices = perform_edge_based_node_sampling_step(coo_list->source, coo_list->destination, fraction);
	printf("\nCollected %d vertices.", sampled_vertices->sampled_vertices_size);

	// Induction step (TODO: re-use device memory from CSR conversion)
	int* d_offsets;
	int* d_indices;
	hipMalloc((void**)&d_offsets, sizeof(int)*(SIZE_VERTICES + 1));
	hipMalloc((void**)&d_indices, sizeof(int)*SIZE_EDGES);
	hipMemcpy(d_indices, h_indices, SIZE_EDGES * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_offsets, h_offsets, sizeof(int)*(SIZE_VERTICES + 1), hipMemcpyHostToDevice);

	int* d_sampled_vertices;
	hipMalloc((void**)&d_sampled_vertices, sizeof(int)*SIZE_VERTICES);
	hipMemcpy(d_sampled_vertices, sampled_vertices->vertices, sizeof(int)*(SIZE_VERTICES), hipMemcpyHostToDevice);

	printf("\nRunning kernel (induction step) with block size %d and thread size %d:", get_block_size(), get_thread_size());
	perform_induction_step <<<get_block_size(), get_thread_size() >> >(d_sampled_vertices, d_offsets, d_indices);

	int h_edge_count;
	hipMemcpyFromSymbol(&h_edge_count, HIP_SYMBOL(d_edge_count), sizeof(int));
	if (h_edge_count >= SIZE_EDGES + 1) {
		printf("overflow error\n"); return;
	}

	printf("\nAmount of edges collected: %d", h_edge_count);
	std::vector<Edge> results(h_edge_count);
	hipMemcpyFromSymbol(&(results[0]), edge_data, h_edge_count * sizeof(Edge));

	write_output_to_file(results, output_path);

	hipFree(d_offsets);
	hipFree(d_indices);
	hipFree(d_sampled_vertices);

	// Cleanup
	free(sampled_vertices->vertices);
	free(sampled_vertices);
	free(coo_list);
	free(h_indices);
	free(h_offsets);
}

/*
Fast conversion to CSR - Using nvGraph for conversion
Modified from: github.com/bmass02/nvGraphExample
*/
void convert_coo_to_csr_format(int* source_vertices, int* target_vertices, int* h_offsets, int* h_indices) {
	printf("\nConverting COO to CSR format.");

	// First setup the COO format from the input (source_vertices and target_vertices array)
	nvgraphHandle_t handle;
	nvgraphGraphDescr_t graph;
	nvgraphCreate(&handle);
	nvgraphCreateGraphDescr(handle, &graph);
	nvgraphCOOTopology32I_t cooTopology = (nvgraphCOOTopology32I_t)malloc(sizeof(struct nvgraphCOOTopology32I_st));
	cooTopology->nedges = SIZE_EDGES;
	cooTopology->nvertices = SIZE_VERTICES;
	cooTopology->tag = NVGRAPH_UNSORTED;

	hipMalloc((void**)&cooTopology->source_indices, SIZE_EDGES * sizeof(int));
	hipMalloc((void**)&cooTopology->destination_indices, SIZE_EDGES * sizeof(int));

	hipMemcpy(cooTopology->source_indices, source_vertices, SIZE_EDGES * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cooTopology->destination_indices, target_vertices, SIZE_EDGES * sizeof(int), hipMemcpyHostToDevice);

	// Edge data (allocated, but not used)
	hipDataType data_type = HIP_R_32F;
	float* d_edge_data;
	float* d_destination_edge_data;
	hipMalloc((void**)&d_edge_data, sizeof(float) * SIZE_EDGES); // Note: only allocate this for 1 float since we don't have any data yet
	hipMalloc((void**)&d_destination_edge_data, sizeof(float) * SIZE_EDGES); // Note: only allocate this for 1 float since we don't have any data yet

	// Convert COO to a CSR format
	nvgraphCSRTopology32I_t csrTopology = (nvgraphCSRTopology32I_t)malloc(sizeof(struct nvgraphCSRTopology32I_st));
	int **d_indices = &(csrTopology->destination_indices);
	int **d_offsets = &(csrTopology->source_offsets);

	hipMalloc((void**)d_indices, SIZE_EDGES * sizeof(int));
	hipMalloc((void**)d_offsets, (SIZE_VERTICES + 1) * sizeof(int));

	check(nvgraphConvertTopology(handle, NVGRAPH_COO_32, cooTopology, d_edge_data, &data_type, NVGRAPH_CSR_32, csrTopology, d_destination_edge_data));

	// Copy data to the host (without edge data)
	hipMemcpy(h_indices, *d_indices, SIZE_EDGES * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_offsets, *d_offsets, (SIZE_VERTICES + 1) * sizeof(int), hipMemcpyDeviceToHost);

	// Clean up (Data allocated on device and both topologies, since we only want to work with indices and offsets for now)
	hipFree(d_indices);
	hipFree(d_offsets);
	hipFree(d_edge_data);
	hipFree(d_destination_edge_data);
	hipFree(cooTopology->destination_indices);
	hipFree(cooTopology->source_indices);
	free(cooTopology);
	free(csrTopology);
}

int get_thread_size() {
	return ((SIZE_VERTICES + 1) > MAX_THREADS) ? MAX_THREADS : SIZE_VERTICES;
}

int get_block_size() {
	return ((SIZE_VERTICES + 1) > MAX_THREADS) ? ((SIZE_VERTICES / MAX_THREADS) + 1) : 1;
}

int calculate_node_sampled_size(float fraction) {
	return int(SIZE_VERTICES * fraction);
}

/*
NOTE: Only reads integer vertices for now (through the 'sscanf' function) and obvious input vertices arrays
*/
void load_graph_from_edge_list_file(int* source_vertices, int* target_vertices, char* file_path) {
	printf("\nLoading graph file from: %s", file_path);

	FILE* file = fopen(file_path, "r");
	char line[256];
	int edge_index = 0;

	while (fgets(line, sizeof(line), file)) {
		if (line[0] == '#') {
			//print_debug_log("\nEscaped a comment.");
			continue;
		}

		// Save source and target vertex (temp)
		int source_vertex;
		int target_vertex;

		sscanf(line, "%d%d\t", &source_vertex, &target_vertex);

		// Add vertices to the source and target arrays, forming an edge accordingly
		source_vertices[edge_index] = source_vertex;
		target_vertices[edge_index] = target_vertex;

		// Increment edge index to add any new edge
		edge_index++;

		//print_debug_log("\nAdded start vertex:", source_vertex);
		//print_debug_log("\nAdded end vertex:", target_vertex);
	}

	fclose(file);
}

COO_List* load_graph_from_edge_list_file_to_coo(std::vector<int>& source_vertices, std::vector<int>& destination_vertices, char* file_path) {
	printf("\nLoading graph file from: %s", file_path);

	std::unordered_map<int, int> map_from_edge_to_coordinate;

	FILE* file = fopen(file_path, "r");
	
	char line[256];

	int current_coordinate = 0;

	while (fgets(line, sizeof(line), file)) {
		if (line[0] == '#') {
			//print_debug_log("\nEscaped a comment.");
			continue;
		}

		// Save source and target vertex (temp)
		int source_vertex;
		int target_vertex;

		sscanf(line, "%d%d\t", &source_vertex, &target_vertex);

		// Add vertices to the source and target arrays, forming an edge accordingly
		current_coordinate = add_vertex_as_coordinate(source_vertices, map_from_edge_to_coordinate, source_vertex, current_coordinate);
		current_coordinate = add_vertex_as_coordinate(destination_vertices, map_from_edge_to_coordinate, target_vertex, current_coordinate);
	}

	COO_List* coo_list = (COO_List*)malloc(sizeof(COO_List));

	source_vertices.reserve(source_vertices.size());
	destination_vertices.reserve(destination_vertices.size());
	coo_list->source = &source_vertices[0];
	coo_list->destination = &destination_vertices[0];

	printf("\nTotal amount of vertices: %zd", map_from_edge_to_coordinate.size());
	printf("\nTotal amount of edges: %zd", source_vertices.size());

	// Print edges
	/*for (int i = 0; i < source_vertices.size(); i++) {
		printf("\n(%d, %d)", coo_list->source[i], coo_list->destination[i]);
	}*/

	fclose(file);

	return coo_list;
}

int add_vertex_as_coordinate(std::vector<int>& vertices_type, std::unordered_map<int, int>& map_from_edge_to_coordinate, int vertex, int coordinate) {
	if (map_from_edge_to_coordinate.count(vertex)) {
		vertices_type.push_back(map_from_edge_to_coordinate.at(vertex));

		return coordinate;
	} else {
		map_from_edge_to_coordinate[vertex] = coordinate;
		vertices_type.push_back(coordinate);
		coordinate++;

		return coordinate;
	}
}

Sampled_Vertices* perform_edge_based_node_sampling_step(int* source_vertices, int* target_vertices, float fraction) {
	printf("\nPerforming edge based node sampling step.\n");

	Sampled_Vertices* sampled_vertices = (Sampled_Vertices*) malloc(sizeof(Sampled_Vertices));

	int amount_total_sampled_vertices = calculate_node_sampled_size(fraction);

	std::random_device seeder;
	std::mt19937 engine(seeder());

	sampled_vertices->vertices = (int*) calloc(SIZE_VERTICES, sizeof(int));
	int collected_amount = 0;

	while (collected_amount < amount_total_sampled_vertices) {
		// Pick a random vertex u
		std::uniform_int_distribution<int> range_edges(0, (SIZE_EDGES-1)); // Don't select the last element in the offset
		int random_edge_index = range_edges(engine);

		// Insert u, v (TODO: extract to method per vertex)
		if (!sampled_vertices->vertices[source_vertices[random_edge_index]]) {
			sampled_vertices->vertices[source_vertices[random_edge_index]] = 1;
			print_debug_log("\nCollected vertex:", source_vertices[random_edge_index]);
			collected_amount++;
		}
		if (!sampled_vertices->vertices[target_vertices[random_edge_index]]) {
			sampled_vertices->vertices[target_vertices[random_edge_index]] = 1;
			print_debug_log("\nCollected vertex:", target_vertices[random_edge_index]);
			collected_amount++;
		}
	}

	sampled_vertices->sampled_vertices_size = collected_amount;

	return sampled_vertices;
}


/*
=======================================================================================
Expanding code
=======================================================================================
*/

void expand_graph(char* input_path, char* output_path, float scaling_factor) {
	std::vector<int> source_vertices;
	std::vector<int> destination_vertices;
	COO_List* coo_list = load_graph_from_edge_list_file_to_coo(source_vertices, destination_vertices, input_path);

	// Convert the COO graph into a CSR format for the in memory GPU representation
	int* h_offsets = (int*)malloc((SIZE_VERTICES + 1) * sizeof(int));
	int* h_indices = (int*)malloc(SIZE_EDGES * sizeof(int));

	convert_coo_to_csr_format(coo_list->source, coo_list->destination, h_offsets, h_indices);

	const int amount_of_sampled_graphs = scaling_factor / DEFAULT_EXPANDING_SAMPLE_SIZE;

	printf("Amount of sampled graphs: %d", amount_of_sampled_graphs);

	Sampled_Vertices** sampled_vertices_per_graph = (Sampled_Vertices**) malloc(sizeof(Sampled_Vertices)*amount_of_sampled_graphs);
	
	int** d_size_edges = (int**) malloc(sizeof(int*)*amount_of_sampled_graphs);
	Edge** d_edge_data_expanding = (Edge**) malloc(sizeof(Edge*)*amount_of_sampled_graphs);

	Sampled_Graph_Version* sampled_graph_version_list = new Sampled_Graph_Version[amount_of_sampled_graphs];
	char current_label = 'a';

	for (int i = 0; i < amount_of_sampled_graphs; i++) {
		sampled_vertices_per_graph[i] = perform_edge_based_node_sampling_step(coo_list->source, coo_list->destination, DEFAULT_EXPANDING_SAMPLE_SIZE);
		printf("\nCollected %d vertices.", sampled_vertices_per_graph[i]->sampled_vertices_size);
		printf("\nDone with node sampling step..");
		// Induction step (TODO: re-use device memory from CSR conversion)
		int* d_offsets;
		int* d_indices;
		hipMalloc((void**)&d_offsets, sizeof(int)*(SIZE_VERTICES + 1));
		hipMalloc((void**)&d_indices, sizeof(int)*SIZE_EDGES);
		hipMemcpy(d_indices, h_indices, SIZE_EDGES * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_offsets, h_offsets, sizeof(int)*(SIZE_VERTICES + 1), hipMemcpyHostToDevice);

		int* d_sampled_vertices;
		hipMalloc((void**)&d_sampled_vertices, sizeof(int)*SIZE_VERTICES);
		hipMemcpy(d_sampled_vertices, sampled_vertices_per_graph[i]->vertices, sizeof(int)*(SIZE_VERTICES), hipMemcpyHostToDevice);

		int* h_size_edges = 0;
		hipMalloc((void**)&d_size_edges[i], sizeof(int));
		hipMemcpy(d_size_edges[i], &h_size_edges, sizeof(int), hipMemcpyHostToDevice);

		hipMalloc((void**)&d_edge_data_expanding[i], sizeof(Edge)*SIZE_EDGES);
		
		hipDeviceSynchronize();

		printf("\nRunning kernel (induction step) with block size %d and thread size %d:", get_block_size(), get_thread_size());
		perform_induction_step_expanding <<<get_block_size(), get_thread_size()>>>(d_sampled_vertices, d_offsets, d_indices, d_edge_data_expanding[i], d_size_edges[i]);
		
		// Edge size
		int h_size_edges_result;
		hipMemcpy(&h_size_edges_result, d_size_edges[i], sizeof(int), hipMemcpyDeviceToHost);

		// Edges
		printf("\nh_size_edges: %d", h_size_edges_result);
		Sampled_Graph_Version* sampled_graph_version = new Sampled_Graph_Version();
		(*sampled_graph_version).edges.resize(h_size_edges_result);

		hipMemcpy(&sampled_graph_version->edges[0], d_edge_data_expanding[i], sizeof(Edge)*(h_size_edges_result), hipMemcpyDeviceToHost);

		// Label
		sampled_graph_version->label = current_label++;

		// Copy data to the sampled version list
		sampled_graph_version_list[i] = (*sampled_graph_version);

		// Cleanup
		delete(sampled_graph_version);
		
		hipFree(d_sampled_vertices);
		hipFree(d_offsets);
		hipFree(d_indices);
		free(sampled_vertices_per_graph[i]->vertices);
		free(sampled_vertices_per_graph[i]);
	}

	free(sampled_vertices_per_graph);
	free(coo_list);
	free(h_indices);
	free(h_offsets);
	
	printf("\nAfter test: %d", sampled_graph_version_list[0].edges.size());

	// For each sampled graph version, copy the data back to the host
	std::vector<Bridge_Edge> bridge_edges;
	link_using_star_topology(sampled_graph_version_list, amount_of_sampled_graphs, bridge_edges);
	
	write_expanded_output_to_file(sampled_graph_version_list, amount_of_sampled_graphs, bridge_edges, output_path);

	// Cleanup
	delete[] sampled_graph_version_list;
	hipFree(d_edge_data_expanding); // Perhaps these cuda allocations can be freed in the for loop..
	hipFree(d_size_edges);
}

void link_using_star_topology(Sampled_Graph_Version* sampled_graph_version_list, int amount_of_sampled_graphs, std::vector<Bridge_Edge>& bridge_edges) {
	/*printf("\nAfter size now 0: %d with label: %c", sampled_graph_version_list[0].edges.size(), sampled_graph_version_list[0].label);
	printf("\nIs there an actual edge here: (%d, %d)", sampled_graph_version_list[0].edges[0].source, sampled_graph_version_list[0].edges[0].destination);
	printf("\nAfter size now 1: %d with label: %c", sampled_graph_version_list[1].edges.size(), sampled_graph_version_list[1].label);
	printf("\nAfter size now 2: %d with label: %c", sampled_graph_version_list[2].edges.size(), sampled_graph_version_list[2].label);
	printf("\nAfter size now 3: %d with label: %c", sampled_graph_version_list[3].edges.size(), sampled_graph_version_list[3].label);*/
	
	// First sampled version will be the graph in the center
	Sampled_Graph_Version center_graph = sampled_graph_version_list[0];
	
	int amount_of_edge_interconnections = 1;
	for (int i = 1; i < amount_of_sampled_graphs; i++) { // Skip the center graph 
		add_edge_interconnection_between_graphs(amount_of_edge_interconnections, &(sampled_graph_version_list[i]), &center_graph, bridge_edges);
	}

	printf("\nCollected a total of %d bridge edges.", bridge_edges.size());
}

/*
-> Probably parallelizable.
-> if(amount_of_edge_interconnections<1) = fraction of the edges/nodes?
*/
void add_edge_interconnection_between_graphs(int amount_of_edge_interconnections, Sampled_Graph_Version* graph_a, Sampled_Graph_Version* graph_b, std::vector<Bridge_Edge>& bridge_edges) {
	printf("\n============================");
	for (int i = 0; i < amount_of_edge_interconnections; i++) {
		int vertex_a = select_random_bridge_vertex(graph_a);
		int vertex_b = select_random_bridge_vertex(graph_b);
		
		// TODO: Extract function
		// Add edge
		Bridge_Edge bridge_edge;
		sprintf(bridge_edge.source, "%c%d", graph_a->label, vertex_a);
		sprintf(bridge_edge.destination, "%c%d", graph_b->label, vertex_b);

		bridge_edges.push_back(bridge_edge);
		printf("\nBridge selection - Selected: (%s, %s)", bridge_edge.source, bridge_edge.destination);
	}
}

// TODO: Add parameter (e.g. Random/high-degree nodes/low-degree nodes)
int select_random_bridge_vertex(Sampled_Graph_Version* graph) {
	// TODO: Move to add_edge_interconnection_between_graphs
	std::random_device seeder;
	std::mt19937 engine(seeder());
	std::uniform_int_distribution<int> range_edges(0, ((*graph).edges.size()) - 1);
	int random_edge_index = range_edges(engine);

	return (*graph).edges[random_edge_index].destination; // Select destination vertex (perhaps make this 50:50?)
}

void write_expanded_output_to_file(Sampled_Graph_Version* sampled_graph_version_list, int amount_of_sampled_graphs, std::vector<Bridge_Edge>& bridge_edges, char* ouput_path) {
	char* file_path = ouput_path;
	FILE *output_file = fopen(file_path, "w");

	if (output_file == NULL) {
		printf("\nError writing results to output file.");
		exit(1);
	}

	// Write sampled graph versions
	for (int i = 0; i < amount_of_sampled_graphs; i++) {
		for (int p = 0; p < sampled_graph_version_list[i].edges.size(); p++) {
			fprintf(output_file, "\n%c%d\t%c%d", sampled_graph_version_list[i].label, sampled_graph_version_list[i].edges[p].source, sampled_graph_version_list[i].label, sampled_graph_version_list[i].edges[p].destination);
		}
	}
	
	for (int i = 0; i < bridge_edges.size(); i++) {
		fprintf(output_file, "\n%s\t%s", bridge_edges[i].source, bridge_edges[i].destination);
	}
		
	fclose(output_file);
}

void write_output_to_file(std::vector<Edge>& results, char* ouput_path) {
	char* file_path = ouput_path;
	FILE *output_file = fopen(file_path, "w");

	if (output_file == NULL) {
		printf("\nError writing results to output file.");
		exit(1);
	}

	for (int i = 0; i < results.size(); i++) {
		fprintf(output_file, "%d\t%d\n", results[i].source, results[i].destination);
	}

	fclose(output_file);
}

void print_coo(int* source_vertices, int* end_vertices) {
	for (int i = 0; i < SIZE_EDGES; i++) {
		printf("\n%d, %d", source_vertices[i], end_vertices[i]);
	}
}

void print_csr(int* h_offsets, int* h_indices) {
	printf("\nRow Offsets (Vertex Table):\n");
	for (int i = 0; i < SIZE_VERTICES + 1; i++) {
		printf("%d, ", h_offsets[i]);
	}

	printf("\nColumn Indices (Edge Table):\n");
	for (int i = 0; i < SIZE_EDGES; i++) {
		printf("%d, ", h_indices[i]);
	}
}

void check(nvgraphStatus_t status) {
	if (status != NVGRAPH_STATUS_SUCCESS) {
		printf("ERROR : %d\n", status);
		exit(0);
	}
}

void print_debug_log(char* message) {
	if (ENABLE_DEBUG_LOG)
		printf("%s", message);
}

void print_debug_log(char* message, int value) {
	if (ENABLE_DEBUG_LOG)
		printf("%s %d", message, value);
}