#include "hip/hip_runtime.h"
/*
NOTE: Run in VS using x64 platform.
*/

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include "kernel.h"
#include <string.h>
#include <nvgraph.h>
#include "hip/device_functions.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <iostream>
#include <unordered_set>
#include <random>

#define SIZE_VERTICES 281903 //20
#define SIZE_EDGES 2312497 //72 NOTE IF YOU AUTOMATE THIS, MAKE SURE TO CHECK WHETHER THE EDGE DATA DEVICE ARRAY STILL WORKS.
#define ENABLE_DEBUG_LOG false

void load_graph_from_edge_list_file(int*, int*, char*);
int get_thread_size();
int calculate_node_sampled_size(float);
int get_block_size();
typedef struct Sampled_Vertices;
Sampled_Vertices* perform_edge_based_node_sampling_step(int*, int*, float);
void print_debug_log(char*);
void print_debug_log(char*, int);
void print_coo(int*, int*);
void print_csr(int*, int*);
void convert_coo_to_csr_format(int*, int*, int*, int*);
void check(nvgraphStatus_t);

typedef struct Sampled_Vertices {
	int* vertices;
	int sampled_vertices_size;
} Sampled_Vertices;

typedef struct {
	int source, destination;
} Edge;

__device__ Edge edge_data[SIZE_EDGES];
__device__ int edge_count = 0;

__device__ int push_edge(Edge &edge) {
	int edge_index = atomicAdd(&edge_count, 1);
	if (edge_index < SIZE_EDGES) {
		edge_data[edge_index] = edge;
		return edge_index;
	} else {
		printf("Maximum edge size threshold reached.");
		return -1;
	}
}

__global__
void perform_induction_step(int* sampled_vertices, int* sampled_vertices_size, int* offsets, int* indices) {
	int neighbor_index_start_offset = blockIdx.x * blockDim.x + threadIdx.x;
	int neighbor_index_end_offset = neighbor_index_start_offset + 1;

	for (int n = offsets[neighbor_index_start_offset]; n < offsets[neighbor_index_end_offset]; n++) {
		bool found_vertex_u = sampled_vertices[neighbor_index_start_offset] != -1;
		bool found_vertex_v = sampled_vertices[indices[n]] != -1;

		if (found_vertex_u && found_vertex_v) {
			//printf("\nAdd edge: (%d,%d).", neighbor_index_start_offset, indices[n]);
			Edge edge;
			edge.source = neighbor_index_start_offset;
			edge.destination = indices[n];
			push_edge(edge);
		}

		found_vertex_u = false;
		found_vertex_v = false;
	}
}

/*
TODO: Allocate the memory on the GPU only when you need it, after collecting the edge-based node step.
*/
int main() {
	int* source_vertices;
	int* target_vertices;
	//char* file_path = "C:\\Users\\AJ\\Documents\\example_graph.txt";
	//char* file_path = "C:\\Users\\AJ\\Desktop\\nvgraphtest\\nvGraphExample-master\\nvGraphExample\\web-Stanford.txt";
	char* file_path = "C:\\Users\\AJ\\Desktop\\nvgraphtest\\nvGraphExample-master\\nvGraphExample\\web-Stanford_large.txt";

	size_t print_size = (sizeof(int) * SIZE_EDGES) + (3000 * sizeof(int));
	hipDeviceSetLimit(hipLimitPrintfFifoSize, print_size);

	source_vertices = (int*)malloc(sizeof(int) * SIZE_EDGES);
	target_vertices = (int*)malloc(sizeof(int) * SIZE_EDGES);

	// Read an input graph into a COO format.
	load_graph_from_edge_list_file(source_vertices, target_vertices, file_path);

	// print_coo(source_vertices, target_vertices);

	// Convert the COO graph into a CSR format for the in memory GPU representation
	int* h_offsets = (int*)malloc((SIZE_VERTICES + 1) * sizeof(int));
	int* h_indices = (int*)malloc(SIZE_EDGES * sizeof(int));

	convert_coo_to_csr_format(source_vertices, target_vertices, h_offsets, h_indices);

	//print_csr(h_offsets, h_indices);

	// Edge based Node Sampling Step
	Sampled_Vertices* sampled_vertices = perform_edge_based_node_sampling_step(source_vertices, target_vertices, 0.5);
	printf("\nCollected %d vertices.", sampled_vertices->sampled_vertices_size);

	// Induction step (TODO: re-use device memory from CSR conversion)
	int* d_offsets;
	int* d_indices;
	hipMalloc((void**)&d_offsets, sizeof(int)*(SIZE_VERTICES + 1));
	hipMalloc((void**)&d_indices, sizeof(int)*SIZE_EDGES);
	hipMemcpy(d_indices, h_indices, SIZE_EDGES * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_offsets, h_offsets, sizeof(int)*(SIZE_VERTICES + 1), hipMemcpyHostToDevice);

	int* d_sampled_vertices;
	hipMalloc((void**)&d_sampled_vertices, sizeof(int)*SIZE_EDGES);
	hipMemcpy(d_sampled_vertices, sampled_vertices->vertices, sizeof(int)*(SIZE_EDGES), hipMemcpyHostToDevice);

	int* d_sampled_vertices_size;
	hipMalloc(&d_sampled_vertices_size, sizeof(int));
	hipMemcpy(d_sampled_vertices_size, &sampled_vertices->sampled_vertices_size, sizeof(int), hipMemcpyHostToDevice);

	printf("\nRunning kernel (induction step) with block size %d and thread size %d:", get_block_size(), get_thread_size());
	perform_induction_step<<<get_block_size(), get_thread_size()>>>(d_sampled_vertices, d_sampled_vertices_size, d_offsets, d_indices);
	
	int amount_collected_edges;
	hipMemcpyFromSymbol(&amount_collected_edges, HIP_SYMBOL(edge_count), sizeof(int));
	if (amount_collected_edges >= SIZE_EDGES + 1) {
		printf("overflow error\n"); return 1; 
	}
	printf("\nAmount of edges collected: %d", amount_collected_edges);
	/*std::vector<Edge> results(dsize);
	hipMemcpyFromSymbol(&(results[0]), edge_data, dsize * sizeof(Edge));
	printf("\nWOOHOO: %d", dsize);
	printf("\nTest: (%d, %d)", results[0].source, results[0].destination);
	*/
	hipFree(d_offsets);
	hipFree(d_indices);
	hipFree(d_sampled_vertices_size);
	hipFree(d_sampled_vertices);

	// Cleanup
	free(sampled_vertices->vertices);
	free(sampled_vertices);
	free(source_vertices);
	free(target_vertices);
	free(h_indices);
	free(h_offsets);

	return 0;
}

/*
Fast conversion to CSR - Using nvGraph for conversion
Modified from: github.com/bmass02/nvGraphExample
*/
void convert_coo_to_csr_format(int* source_vertices, int* target_vertices, int* h_offsets, int* h_indices) {
	printf("\nConverting COO to CSR format.");

	// First setup the COO format from the input (source_vertices and target_vertices array)
	nvgraphHandle_t handle;
	nvgraphGraphDescr_t graph;
	nvgraphCreate(&handle);
	nvgraphCreateGraphDescr(handle, &graph);
	nvgraphCOOTopology32I_t cooTopology = (nvgraphCOOTopology32I_t)malloc(sizeof(struct nvgraphCOOTopology32I_st));
	cooTopology->nedges = SIZE_EDGES;
	cooTopology->nvertices = SIZE_VERTICES;
	cooTopology->tag = NVGRAPH_UNSORTED;

	hipMalloc((void**)&cooTopology->source_indices, SIZE_EDGES * sizeof(int));
	hipMalloc((void**)&cooTopology->destination_indices, SIZE_EDGES * sizeof(int));

	hipMemcpy(cooTopology->source_indices, source_vertices, SIZE_EDGES * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cooTopology->destination_indices, target_vertices, SIZE_EDGES * sizeof(int), hipMemcpyHostToDevice);

	// Edge data (allocated, but not used)
	hipDataType data_type = HIP_R_32F;
	float* d_edge_data;
	float* d_destination_edge_data;
	hipMalloc((void**)&d_edge_data, sizeof(float) * SIZE_EDGES); // Note: only allocate this for 1 float since we don't have any data yet
	hipMalloc((void**)&d_destination_edge_data, sizeof(float) * SIZE_EDGES); // Note: only allocate this for 1 float since we don't have any data yet

	// Convert COO to a CSR format
	nvgraphCSRTopology32I_t csrTopology = (nvgraphCSRTopology32I_t)malloc(sizeof(struct nvgraphCSRTopology32I_st));
	int **d_indices = &(csrTopology->destination_indices);
	int **d_offsets = &(csrTopology->source_offsets);

	hipMalloc((void**)d_indices, SIZE_EDGES * sizeof(int));
	hipMalloc((void**)d_offsets, (SIZE_VERTICES + 1) * sizeof(int));

	check(nvgraphConvertTopology(handle, NVGRAPH_COO_32, cooTopology, d_edge_data, &data_type, NVGRAPH_CSR_32, csrTopology, d_destination_edge_data));

	// Copy data to the host (without edge data)
	hipMemcpy(h_indices, *d_indices, SIZE_EDGES * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_offsets, *d_offsets, (SIZE_VERTICES + 1) * sizeof(int), hipMemcpyDeviceToHost);

	// Clean up (Data allocated on device and both topologies, since we only want to work with indices and offsets for now)
	hipFree(d_indices);
	hipFree(d_offsets);
	hipFree(d_edge_data);
	hipFree(d_destination_edge_data);
	hipFree(cooTopology->destination_indices);
	hipFree(cooTopology->source_indices);
	free(cooTopology);
	free(csrTopology);
}

int get_thread_size() {
	return ((SIZE_VERTICES + 1) > 1024) ? 1024 : SIZE_VERTICES;
}

int get_block_size() {
	return ((SIZE_VERTICES + 1) > 1024) ? ((SIZE_VERTICES / 1024) + 1) : 1;
}

int calculate_node_sampled_size(float fraction) {
	return int(SIZE_VERTICES * fraction);
}

/*
NOTE: Only reads integer vertices for now (through the 'sscanf' function) and obvious input vertices arrays
*/
void load_graph_from_edge_list_file(int* source_vertices, int* target_vertices, char* file_path) {
	printf("\nLoading graph file from: %s", file_path);

	FILE* file = fopen(file_path, "r");
	char line[256];
	int edge_index = 0;

	while (fgets(line, sizeof(line), file)) {
		if (line[0] == '#') {
			//log("\nEscaped a comment.");
			continue;
		}

		// Save source and target vertex (temp)
		int source_vertex;
		int target_vertex;

		sscanf(line, "%d%d\t", &source_vertex, &target_vertex);

		// Add vertices to the source and target arrays, forming an edge accordingly
		source_vertices[edge_index] = source_vertex;
		target_vertices[edge_index] = target_vertex;

		// Increment edge index to add any new edge
		edge_index++;

		//log("\nAdded start vertex:", source_vertex);
		//log("\nAdded end vertex:", target_vertex);
	}

	fclose(file);
}

Sampled_Vertices* perform_edge_based_node_sampling_step(int* source_vertices, int* target_vertices, float fraction) {
	printf("\nPerforming edge based node sampling step.");
	
	Sampled_Vertices* sampled_vertices = (Sampled_Vertices*) malloc(sizeof(Sampled_Vertices));

	int amount_total_sampled_vertices = calculate_node_sampled_size(fraction);

	std::random_device seeder;
	std::mt19937 engine(seeder());

	sampled_vertices->vertices = (int*)calloc(SIZE_EDGES, sizeof(int));
	int collected_amount = 0;

	// TODO: memcpy
	for (int x = 0; x < SIZE_EDGES; x++) {
		sampled_vertices->vertices[x] = -1;
	}

	while (collected_amount <= amount_total_sampled_vertices) {
		// Pick a random vertex u
		std::uniform_int_distribution<int> range_edges(0, (SIZE_EDGES - 1)); // Don't select the last element in the offset
		int random_edge_index = range_edges(engine);

		// Insert u, v 
		if (sampled_vertices->vertices[source_vertices[random_edge_index]] == -1) {
			sampled_vertices->vertices[source_vertices[random_edge_index]] = source_vertices[random_edge_index];
			//log("\nCollected vertex: %d", source_vertices[random_edge_index]);
			collected_amount++;
		}
		if (sampled_vertices->vertices[target_vertices[random_edge_index]] == -1) {
			sampled_vertices->vertices[target_vertices[random_edge_index]] = target_vertices[random_edge_index];
			//log("\nCollected vertex: %d", target_vertices[random_edge_index]);
			collected_amount++;
		}
	}

	sampled_vertices->sampled_vertices_size = collected_amount;

	return sampled_vertices;
}

void print_coo(int* source_vertices, int* end_vertices) {
	for (int i = 0; i < SIZE_EDGES; i++) {
		printf("\n%d, %d", source_vertices[i], end_vertices[i]);
	}
}

void print_csr(int* h_offsets, int* h_indices) {
	printf("\nRow Offsets (Vertex Table):\n");
	for (int i = 0; i < SIZE_VERTICES + 1; i++) {
		printf("%d, ", h_offsets[i]);
	}

	printf("\nColumn Indices (Edge Table):\n");
	for (int i = 0; i < SIZE_EDGES; i++) {
		printf("%d, ", h_indices[i]);
	}
}

void check(nvgraphStatus_t status) {
	if (status != NVGRAPH_STATUS_SUCCESS) {
		printf("ERROR : %d\n", status);
		exit(0);
	}
}

void print_debug_log(char* message) {
	if (ENABLE_DEBUG_LOG)
		printf("%s", message);
}

void print_debug_log(char* message, int value) {
	if (ENABLE_DEBUG_LOG)
		printf("%s %d", message, value);
}